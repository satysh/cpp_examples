#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N (1024*1024)

_global_ void kernel (float* dA)
{
    int idx = blockIdx.x * blockDim.x + threadId.x;
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    dA[idx] = sinf(sqrtf(x));
}

int main(int argc, char const *argv[])
{
    float* dA;
    float* hA;

    hA = (float*)malloc(N * sizeof(float));

    hipError_t errMem;
    errMem = hipMalloc((void**)&dA, N * sizeof(float));
    if (errMem != hipSuccess)
    {
        fprintf(stderr, "Can not allocate GPU memory: %s\n", hipGetErrorString(errMem));
        return 1;
    }

    kernel <<< N/512, 512 >>> (dA);
    hipError_t err;
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Can not launch CUDA kernel: %s\n", hipGetErrorString(err));
        return 1;
    }

    hipMemcpy(hA, dA, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int idx = 0; idx < N; idx++)
    {
        printf("a[%d]=%.5f\n", idx, hA[idx]);
    }

    free(hA);
    hipFree(dA);
    return 0;
}
